#include "hip/hip_runtime.h"
#include "quark/util/math_functions.h"

namespace quark {

template <typename T>
__global__ void EltwiseProduct(int64 n, const T* alpha, const T* a, const T* b, T* c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    c[tid] = alpha * b[tid] * c[tid];
  }
}

template <typename T>
void quark_gpu_eltwise_prod(hipStream_t stream, const T* alpha, const GpuTensor<T> &a,
    const GpuTensor<T>& b, GpuTensor<T>* c) {
  int64 n = Prod(a.shape());

  int num_thread = QUARK_CUDA_BLOCK_SIZE;
  int64 num_block = QUARK_GET_NUM_BLOCK(num_thread);
  EltwiseProduct<<<num_block, num_thread, 0, stream>>>(n, alpha, a.data(),
      b.data(), c->mutable_data());
}

} // namespace quark
